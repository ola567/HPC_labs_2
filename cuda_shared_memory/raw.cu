#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

__host__ void errorexit(const char *s)
{
    printf("\n%s\n", s);
    exit(EXIT_FAILURE);
}

__global__ void computeSum(int *randomNumbers, unsigned long long *resultSumDevice, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N)
    {
        atomicAdd(resultSumDevice, (unsigned long long)randomNumbers[idx]);
    }
}

void generateRandomNumbers(int *arr, int N, int A, int B)
{
    srand(time(NULL));
    for (int i = 0; i < N; i++)
    {
        arr[i] = A + rand() % (B - A + 1);
    }
}

int main(int argc, char **argv)
{
    int threadsinblock = 1024;
    int blocksingrid;

    int N, A, B;
    float milliseconds = 0;

    // Get user input
    printf("Enter number of elements: \n");
    scanf("%d", &N);
    printf("Enter A value (start range): \n");
    scanf("%d", &A);
    printf("Enter B value (end range): \n");
    scanf("%d", &B);

    // Allocate host memory for random numbers
    int *randomNumbers = (int *)malloc(N * sizeof(int));
    if (randomNumbers == NULL)
    {
        printf("Memory allocation failed.\n");
        return 1;
    }

    // Generate random numbers
    generateRandomNumbers(randomNumbers, N, A, B);
    blocksingrid = (N + threadsinblock - 1) / threadsinblock; // Calculate number of blocks
    printf("The kernel will run with: %d blocks\n", blocksingrid);

    // Device memory pointers
    unsigned long long *resultSumDevice;
    int *randomNumbersDevice;
    unsigned long long resultSum = 0;

    // CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // Allocate device memory
    if (hipMalloc((void **)&randomNumbersDevice, N * sizeof(int)) != hipSuccess)
    {
        errorexit("Failed to allocate device memory for randomNumbers");
    }
    if (hipMalloc((void **)&resultSumDevice, sizeof(unsigned long long)) != hipSuccess)
    {
        errorexit("Failed to allocate device memory for resultSumDevice");
    }

    // Copy data to device
    hipMemcpy(randomNumbersDevice, randomNumbers, N * sizeof(int), hipMemcpyHostToDevice);

    // Initialize device result sum to 0
    hipMemset(resultSumDevice, 0, sizeof(unsigned long long));

    // Launch kernel
    computeSum<<<blocksingrid, threadsinblock>>>(randomNumbersDevice, resultSumDevice, N);

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        errorexit(hipGetErrorString(err));
    }

    // Copy result back to host
    hipMemcpy(&resultSum, resultSumDevice, sizeof(unsigned long long), hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    // Calculate and print results
    // for (int i = 0; i < N; i++){
    //     printf("%d, ", randomNumbers[i]);
    // }
    // printf("\n");
    double average = (double)resultSum / N;
    printf("Average: %.2f\n", average);

    // Print execution time
    printf("Kernel execution time: %.3f ms\n", milliseconds);

    // Free allocated memory
    free(randomNumbers);
    hipFree(randomNumbersDevice);
    hipFree(resultSumDevice);

    return 0;
}
